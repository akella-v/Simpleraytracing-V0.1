
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include<hip/hip_runtime_api.h>


#define W  1200 /*****canvas Width*****/
#define H 1200 /*****canvas Height*****/
int C[100000], address[100000][2]; //buffers to store data for CUDA application
int count = 0;

struct vector {
    int x,y;
};

struct ray {
    struct vector origin;
    struct vector dest;
};



///////////////////////////////////// FUNCTIONS CODE ////////////////////////////////////////
void canvas_init (int canvas[H][W]){
    int x,y;

    //for each pixel
    //initializing the canvas to be black
    for( y =0; y < H; y++){
        for(x = 0; x < W; x++){
            canvas[y][x] = 0;
        }
    }

}


void circle(int x, int y ,int canvas[H][W]){
    // creates a circle on the blank canvas with a radius of 9 pixels
    int garbage = 0;
    if ( ((x - 9)<=0) || ((x + 9)>= W) )
    garbage =+ 1;//printf("Circle will not fit the width for given (%d,%d) coordinates",x,y);
    else
    {
        if( ((y - 9)<=0) || ((y + 9)>= H) )
        garbage = 0;//printf("Circle will not fit the Height for given (%d,%d) coordinates",x,y);
        else
        {
            //half circle to the right 0 to 90
            canvas[y][x+9] = 150;
            canvas[y-1][x+9] = 150;
            canvas[y-2][x+9] = 150;
            canvas[y-3][x+9-1] = 150;
            canvas[y-4][x+9-1] = 150;
            canvas[y-5][x+9-2] = 150;
            canvas[y-6][x+9-2] = 150;
            canvas[y-6][x+9-3] = 150;
            canvas[y-7][x+9-3] = 150;
            canvas[y-7][x+9-4] = 150;
            canvas[y-8][x+9-5] = 150;
            canvas[y-8][x+9-6] = 150;
            canvas[y-9][x+9-7] = 150;
            canvas[y-9][x+9-8] = 150;
            canvas[y-9][x] = 150;
            //////////////////////////////

            //half circle to the left 90 to 180
            canvas[y][x+9] = 150;
            canvas[y+1][x+9] = 150;
            canvas[y+2][x+9] = 150;
            canvas[y+3][x+9-1] = 150;
            canvas[y+4][x+9-1] = 150;
            canvas[y+5][x+9-2] = 150;
            canvas[y+6][x+9-2] = 150;
            canvas[y+6][x+9-3] = 150;
            canvas[y+7][x+9-3] = 150;
            canvas[y+7][x+9-4] = 150;
            canvas[y+8][x+9-5] = 150;
            canvas[y+8][x+9-6] = 150;
            canvas[y+9][x+9-7] = 150;
            canvas[y+9][x+9-8] = 150;
            canvas[y+9][x] = 150;
            //////////////////////////////

            //half circle to the left 270 to 360
            canvas[y][x-9] = 150;
            canvas[y-1][x-9] = 150;
            canvas[y-2][x-9] = 150;
            canvas[y-3][x-9+1] = 150;
            canvas[y-4][x-9+1] = 150;
            canvas[y-5][x-9+2] = 150;
            canvas[y-6][x-9+2] = 150;
            canvas[y-6][x-9+3] = 150;
            canvas[y-7][x-9+3] = 150;
            canvas[y-7][x-9+4] = 150;
            canvas[y-8][x-9+5] = 150;
            canvas[y-8][x-9+6] = 150;
            canvas[y-9][x-9+7] = 150;
            canvas[y-9][x-9+8] = 150;
            canvas[y-9][x] = 150;
            //////////////////////////////

            //half circle to the left 180 to 270
            canvas[y][x-9] = 150;
            canvas[y+1][x-9] = 150;
            canvas[y+2][x-9] = 150;
            canvas[y+3][x-9+1] = 150;
            canvas[y+4][x-9+1] = 150;
            canvas[y+5][x-9+2] = 150;
            canvas[y+6][x-9+2] = 150;
            canvas[y+6][x-9+3] = 150;
            canvas[y+7][x-9+3] = 150;
            canvas[y+7][x-9+4] = 150;
            canvas[y+8][x-9+5] = 150;
            canvas[y+8][x-9+6] = 150;
            canvas[y+9][x-9+7] = 150;
            canvas[y+9][x-9+8] = 150;
            canvas[y+9][x] = 150;
            //////////////////////////////
        }
    }
}


void brightness(int a, int b, int canvas[H][W]){
    if(canvas[a][b] <= 250){
        C[count] = canvas[a][b]; address[count][0] = a; address [count][1] = b;count++;
    }
}



void raytracing (struct ray temp_ray, int canvas[H][W]){
  //creating a equation
  int a,b;

  for(a = 0;a < H; a++){
      for(b = 0; b < W; b++){

          //detection of intersection
          // (y - y1) = m(x - x1)
          // m =(y2 - y1)/(x2 -x1)
          /////////////////////////
          int line_equation;
          //check for other points beside them
          line_equation = (temp_ray.dest.y - temp_ray.origin.y)*(b - temp_ray.origin.x)-(temp_ray.dest.x - temp_ray.origin.x)* (a - temp_ray.origin.y);

          if(line_equation == 0)
          {
              /*if(canvas[a][b] < 50) //if u want the ray to be visible
              canvas[a][b] =+ 50;*/
              for(int i = 0; i<9; i++){
                  for(int j = 0; j<9; j++){
                  if(canvas[a -4 +j][b -4 + i] > 100)
                  brightness(a -4 + j, b -4 + i, canvas);
                  }
              }
          }

      }
  }
  ///////////////////////////////////////////////////////////////////////////////////////
}




void printcanvas (int canvas[H][W]){
    int x,y;
    //for each pixel
    for( y =0; y < H; y++){
      for(x = 0; x < W; x++){
          //priting the image
          printf("%d, ",canvas[x][y]);
      }
      printf("\n");
    }
}

///////////// Forward declaration of the device multiplication function/////////////////////
__global__ void VectorAddCuda(int*, int);



__global__ void VectorAddCuda(int* C, int width)
{
 // Block index
int bx = blockIdx.x;
 // Thread index
int tx = threadIdx.x;
 // Index of the vector  processed by the block

int index = bx*blockDim.x + tx;

 // each thread adds corresponding  elements  matrix C
 C[index] = C[index] + 100;

}




///////////////////////////////////// MAIN CODE ////////////////////////////////////////
int main (int argc, char *argv[]){

  //parameters initialization
  int canvas[H][W];
  srand(time(0));
  struct ray ray[10]; //for creating a ray out of two points origin and dest
      ray[0].origin.x = H;
      ray[0].origin.y = H;
      ray[0].dest.x = 0;
      ray[0].dest.y = 0;
  for(int z = 1; z <10; z++)
  {
      ray[z].origin.x = H - z*100;
      ray[z].origin.y = H;
      ray[z].dest.x = z*100;
      ray[z].dest.y = 0;
  }

  //initializing the canvas to be black
  canvas_init(canvas);

  ///creating circles on the canvas

  for(int i = 0; i <200; i++){
      int number = rand()% H;
      int number2 = rand()% H;
      circle (number + 1,number2,canvas);
  }


  //raytracing
  for(int z = 0;z < 10; z++){ ///loop for multiple rays
      raytracing (ray[z],canvas);
  }
/////////////////////////////////////////////////////////CUDA PART OF THE PROGRAM//////////////////////////////////
  //incase you want to see manual timestamps then remove //*** after here and comment out the printcanvas at the end.//

int N = count;
int Cserial[N];
// initialize the input matrices

srand(time(NULL));   // Initialization, should only be called once.


clock_t before_init = clock();
for (int i = 0; i < N; i++){
	Cserial[i] = C[i];
}

clock_t after_init = clock();

int width = N;


hipProfilerStart();
int size=width*sizeof(int);
 // Allocate device memory for  C matrix
int* d_C;
hipError_t err = hipMalloc((void**)&d_C, size);
//***printf("CUDA malloc d_C: %s\n",cudaGetErrorString(err));


 // Copy Matrix data from host memory to device memory
hipEvent_t start_memcpyh2d,stop_memcpyh2d;
hipEventCreate(&start_memcpyh2d);
hipEventCreate(&stop_memcpyh2d);

hipEventRecord(start_memcpyh2d);
err = hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
//***printf("CUDA Memcpy C->Cd: %s\n",cudaGetErrorString(err));
hipEventRecord(stop_memcpyh2d);
float ms1 = 0;

hipEventElapsedTime(&ms1,start_memcpyh2d,stop_memcpyh2d);

//***printf("Time of the MEMCPY of %d bytes: %2.3f ms\n",size,ms1);


hipEvent_t start_kernel,stop_kernel;
hipEventCreate(&start_kernel);
hipEventCreate(&stop_kernel);
// Specify the execution configuration

//dim3 dimBlock(,);
//dim3 dimGrid(,);

// Launch the CUDA kernel on the device


hipEventRecord(start_kernel);
VectorAddCuda<<<N/1000, 1000>>>(d_C,width);
hipEventRecord(stop_kernel);
hipError_t errk = hipDeviceSynchronize();
hipEventSynchronize(stop_kernel);


//***printf("CUDA kernel launch: %s\n",cudaGetErrorString(errk));
 // Read C from the device
hipEvent_t start_memcpyd2h,stop_memcpyd2h;
hipEventCreate(&start_memcpyd2h);
hipEventCreate(&stop_memcpyd2h);
hipEventRecord(start_memcpyd2h);
err = hipMemcpy(&C, d_C, size, hipMemcpyDeviceToHost);
hipEventRecord(stop_memcpyd2h);
//***printf("CUDA Memcpy d_C->C: %s\n",cudaGetErrorString(err));
float ms2 = 0;
hipEventElapsedTime(&ms2,start_memcpyd2h,stop_memcpyd2h);
//***printf("Time of the MEMCPY of %d bytes : %2.3f ms\n",size,ms2);
 // Free device memory
hipFree(d_C);

float milliseconds = 0;
hipEventElapsedTime(&milliseconds,start_kernel,stop_kernel);
//***printf("Time to complete CUDA Vector addition kernel of %d size: %2.3f ms\n",width,milliseconds);

hipProfilerStop();

clock_t before_serial = clock();
for (int k=0; k < N; k++) {
	Cserial[k] += 50;
}


clock_t after_serial = clock();

float serial_totaltime = ((after_serial-before_serial)*1000)/CLOCKS_PER_SEC;
float speedup = serial_totaltime/milliseconds;
//***printf("Execution time for initialization(msec) = %d\n",(((after_init-before_init)*1000)/CLOCKS_PER_SEC));
//***printf("Execution time for CUDA Vector addition(msec)= %2.3f\n",milliseconds);
//***printf("Execution time for serial execution(msec)  %d\n",(((after_serial-after_init)*1000)/CLOCKS_PER_SEC));
//***printf("Speedup obtained by CUDA for %d size Vector addition: %f\n",N,speedup);
//***printf("Ccount = %d, address[count][1] = %d \n", count, address[count][1]);
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



  //printing the canvas

  for(count;count>=0;count--){
  canvas[address[count][0]][address[count][1]] =+ 100;
  }

  printcanvas (canvas);

  return 0;
}
